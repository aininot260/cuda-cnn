#include "global_gpu.cuh"

hipStream_t *stream;

__constant__ float _alpha;
__constant__ int _minibatch;
__constant__ int _epochs;

__device__ int _correct_cnt;
__device__ float _avg_error;

__device__ float _train_image[TRAIN_NUM][ROW][COL];
__device__ int _train_label[TRAIN_NUM];
__device__ float _test_image[TEST_NUM][ROW][COL];
__device__ int _test_label[TEST_NUM];

__device__ float _conv_w[CONV_W_NUM][CONV_W_SIZE][CONV_W_SIZE];
__device__ float _conv_b[CONV_W_NUM];
__device__ float _fc1_b[FC1_SIZE];
__device__ float _fc1_w[FC1_SIZE][CONV_W_NUM][POOL_SIZE][POOL_SIZE]; 
__device__ float _fc2_b[FC2_SIZE];
__device__ float _fc2_w[FC2_SIZE][FC1_SIZE];

__device__ float _input[N_STREAM][ROW][COL];
__device__ float _conv_z[N_STREAM][CONV_W_NUM][CONV_SIZE][CONV_SIZE];
__device__ float _conv_a[N_STREAM][CONV_W_NUM][CONV_SIZE][CONV_SIZE];
__device__ int _pool_pos[N_STREAM][CONV_W_NUM][POOL_SIZE][POOL_SIZE];
__device__ float _pool[N_STREAM][CONV_W_NUM][POOL_SIZE][POOL_SIZE];
__device__ float _fc1_z[N_STREAM][FC1_SIZE];
__device__ float _fc1_a[N_STREAM][FC1_SIZE];
__device__ float _fc2_z[N_STREAM][FC2_SIZE];
__device__ float _fc2_a[N_STREAM][FC2_SIZE];
__device__ float _output[N_STREAM][FC2_SIZE];
__device__ int _answer[N_STREAM][FC2_SIZE];

__device__ float _conv_dw[N_STREAM][CONV_W_NUM][CONV_W_SIZE][CONV_W_SIZE];
__device__ float _conv_db[N_STREAM][CONV_W_NUM];
__device__ float _fc1_db[N_STREAM][FC1_SIZE];
__device__ float _fc1_dw[N_STREAM][FC1_SIZE][CONV_W_NUM][POOL_SIZE][POOL_SIZE];
__device__ float _fc2_db[N_STREAM][FC2_SIZE];
__device__ float _fc2_dw[N_STREAM][FC2_SIZE][FC1_SIZE];
__device__ float _C[N_STREAM][FC2_SIZE];
__device__ float _fc2_delta[N_STREAM][FC2_SIZE];
__device__ float _fc1_delta[N_STREAM][FC1_SIZE];
__device__ float _conv_sigma_delta[N_STREAM][CONV_W_NUM];
__device__ float _conv_delta[N_STREAM][CONV_W_NUM][POOL_SIZE][POOL_SIZE];

__device__ int tmp;