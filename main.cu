#include "hip/hip_runtime.h"
#include "stdio.h"

#include "io.cuh"
#include "fp.cuh"
#include "bp.cuh"
#include "init.cuh"
#include "utils.cuh"
#include "global.cuh"

#include "fp_gpu.cuh"
#include "bp_gpu.cuh"
#include "global_gpu.cuh"

float max_acc;
clock_t t;

int main(int argc,char *argv[])
{
    printf("====== aininot260 gh@ysucloud.com ======\n");
    printf("         Processor used : %s\n",argv[1]);
    printf("         Learning rate  : %.2f\n",alpha);
    printf("         Epochs         : %d\n",epochs);
    printf("         Batch size     : %d\n",minibatch);
    printf("========================================\n");
    printf("\n");

    load_data();
    init_params();

    if(strcmp(argv[1],"CPU")==0)
    {
        for(int i=1;i<=epochs;i++)
        {
            t=clock();
            correct_cnt=0;
            avg_error=0;
    
            for(int j=0;j<TRAIN_NUM;j++)
            {
                set_input(j,train_image);
                input_conv();
                conv_pool();
                pool_fc1();
                fc1_fc2();
                set_answer(j,train_label);
                check_answer(correct_cnt);
                get_error(avg_error);
    
                update_fc2_b();
                update_fc2_w();
                update_fc1_b();
                update_fc1_w();
                update_conv_b();
                update_conv_w();
                if((j+1)%minibatch==0)
                    assign_grads();
            }
            printf("Training  Time spent : %.0fs Image count : %d Accuracy : %0.4f%% Error : %0.4f%% Epoch : %d \n",floor(((float)(clock()-t))/CLOCKS_PER_SEC),TRAIN_NUM,((float)correct_cnt/TRAIN_NUM)*100,(avg_error/TRAIN_NUM)*100,i);
        
            correct_cnt=0;
            avg_error=0;
    
            for(int j=0;j<TEST_NUM;j++)
            {
                set_input(j,test_image);
                input_conv();
                conv_pool();
                pool_fc1();
                fc1_fc2();
                set_answer(j,test_label);
                check_answer(correct_cnt);
                get_error(avg_error);
            }
            printf("Testing   Time spent : %.0fs Image count : %d Accuracy : %0.4f%% Error : %0.4f%% \n",floor(((float)(clock()-t))/CLOCKS_PER_SEC),TEST_NUM,((float)correct_cnt/TEST_NUM)*100,(avg_error/TEST_NUM)*100);
        
            if((float)correct_cnt/TEST_NUM*100>max_acc)
            {
                max_acc=(float)correct_cnt/TEST_NUM*100;
                export_params();
                printf("The new model has been exported. Accuracy has reached to %0.5f%%\n\n",max_acc);
            }
            else
            {
                alpha=alpha-(alpha/3);
                printf("Learning rate has been reduced to %f\n\n",alpha);
            }
        }
    }
    else if(strcmp(argv[1],"GPU")==0)
    {
        initDevice(0);
        int n_stream=N_STREAM;

        stream=(hipStream_t*)malloc(n_stream*sizeof(hipStream_t));
        for(int i=0;i<n_stream;i++)
            hipStreamCreateWithFlags(&stream[i],hipStreamNonBlocking);

        for(int i=1;i<=epochs;i++)
        {
            t=clock();
            correct_cnt=0;
            avg_error=0;
    
            for(int j=0;j<TRAIN_NUM;j++)
            {
                fp_conv_pool_gpu(j,1);
                fp_fc_answer_gpu(j,1);
    
                bp_update_gpu(j);
                if((j+1)%minibatch==0)
                    bp_assign_grads_gpu(j);
            }
    
            hipDeviceSynchronize();
            printf("Training  Time spent : %.0fs Image count : %d Accuracy : %0.4f%% Error : %0.4f%% Epoch : %d \n",floor(((float)(clock()-t))/CLOCKS_PER_SEC),TRAIN_NUM,((float)correct_cnt/TRAIN_NUM)*100,(avg_error/TRAIN_NUM)*100,i);
    
            correct_cnt=0;
            avg_error=0;
    
            for(int j=0;j<TEST_NUM;j++)
            {
                fp_conv_pool_gpu(j,0);
                fp_fc_answer_gpu(j,0);
            }
    
            hipDeviceSynchronize();
            printf("Testing   Time spent : %.0fs Image count : %d Accuracy : %0.4f%% Error : %0.4f%% \n",floor(((float)(clock()-t))/CLOCKS_PER_SEC),TEST_NUM,((float)correct_cnt/TEST_NUM)*100,(avg_error/TEST_NUM)*100);
        
            if((float)correct_cnt/TEST_NUM*100>max_acc)
            {
                max_acc=(float)correct_cnt/TEST_NUM*100;
                export_params();
                printf("The new model has been exported. Accuracy has reached to %0.5f%%\n\n",max_acc);
            }
            else
            {
                alpha=alpha-(alpha/10);
                printf("Learning rate has been reduced to %f\n\n",alpha);
            }
        }

        for(int i=0;i<n_stream;i++)
            hipStreamDestroy(stream[i]);
        free(stream);
    }
    else
    {
        printf("The parameter can only be GPU or CPU!\n");
        return 0;
    }
    return 0;
}